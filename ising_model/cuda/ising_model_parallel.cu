// ################################################################################################## //
//                                     ising_model.cu                                                 //
// ################################################################################################## //
/*
////////////////////////////////////////////////////////////////////////////////////////////////////////
// Johnson Liu ( j_liu21@u.pacific.edu ) 20210422                                                     //
// University of the Pacific                                                                          //
// CA USA                                                                                             //
////////////////////////////////////////////////////////////////////////////////////////////////////////
########################################################################################################
--------------------------------------------------------------------------------------------------------
Functions of the code: ---------------------------------------------------------------------------------
    1)

--------------------------------------------------------------------------------------------------------
Notes: -------------------------------------------------------------------------------------------------
    1)

--------------------------------------------------------------------------------------------------------
Input: -------------------------------------------------------------------------------------------------
    1)

--------------------------------------------------------------------------------------------------------
Output: ------------------------------------------------------------------------------------------------
    A) To screen:
        1)

    B) Printed to ... :
        1)
--------------------------------------------------------------------------------------------------------
########################################################################################################
--------------------------------------------------------------------------------------------------------
1st Update

--------------------------------------------------------------------------------------------------------

########################################################################################################
*/
// =====================================================================================================
// Import libraries to use various functions.
#include <time.h>                   // For measuring runtimes.
#include <sys/time.h>               // For measuring runtimes.
#include <stdlib.h>                 /* srand, rand */
#include <stdio.h>                  // For interaction with console.
#include <iostream>                 // For printing to screen (std::cout, std::endl).
#include <fstream>
#include <hip/hip_runtime.h>                   // For CUDA parallelization on GPU.
// =====================================================================================================


void initialize_grid( int *grid, int length )
{
    int i, j, index;
    float r;
    for ( i = 0; i < length; i++ )
    {
        for ( j = 0; j < length; j++ )
        {
            r = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
            index = ( length * i ) + j;

            if ( r <= 0.5 )
            {
                (grid)[ index ] = -1;
            }
            else
            {
                (grid)[ index ] = 1;
            }
        }
    }
}

void initialize_x1_grid( float a, float q, float r, float m, float *x1_grid, int length )
{
    int i, j, index;
    float x1 = pow( 5.5, 13 );

    for ( i = 0; i < length; i++ )
    {
        for ( j = 0; j < length; j++ )
        {
            index = length * i + j;

            x1 = a * fmod( x1, q ) - ( r * x1 ) / q;

            if ( x1 < 0 )
            {
                x1 += m;
            }

            index = ( length * i ) + j;

            x1_grid[ index ] = x1;
        }
    }
}

void print_grid( int *grid, int length, int t )
{
    int i, j, index, spin;

    int n_zero = 5;
    std::string num_string = std::to_string( t );

    int string_length = num_string.length();

    for ( int k = 0; k < n_zero - string_length; k++ )
    {
        num_string = "0" + num_string;
    }

    std::string filename = "grid_t_" + num_string + ".out";
    std::ofstream outfile ( filename );

    for ( i = 0; i < length; i++)
    {
        for ( j = 0; j < length; j++ )
        {
            index = ( length * i ) + j;
            spin = (grid)[ index ];

            if ( spin == 1)
            {
                // std::cout << " " << 1 <<  ", ";
                outfile << " " << 1 <<  ", ";
            }
            else
            {
                // std::cout << -1 <<  ", ";
                outfile << -1 <<  ", ";
            }
        }
        // std::cout << std::endl;
        outfile << std::endl;
    }

    outfile.close();
}

void print_other_grid( float *other_grid, int length, int t, std::string type )
{
    int i, j, index;
    float other;

    int n_zero = 5;
    std::string num_string = std::to_string( t );

    int string_length = num_string.length();

    for ( int k = 0; k < n_zero - string_length; k++ )
    {
        num_string = "0" + num_string;
    }

    std::string filename = type + "_grid_t_" + num_string + ".out";
    std::ofstream outfile ( filename );

    for ( i = 0; i < length; i++)
    {
        for ( j = 0; j < length; j++ )
        {
            index = ( length * i ) + j;
            other = (other_grid)[ index ];

            outfile << other << ", ";
        }
        // std::cout << std::endl;
        outfile << std::endl;
    }

    outfile.close();
}

void determine_ij( int i, int j, int length, int *ij )
{
    int i_up, i_down, j_left, j_right;

    if ( i == 0 )
    {
        i_up = 1;
        i_down = length - 1;
    }
    else if ( i == length - 1)
    {
        i_up = 0;
        i_down = i - 1;
    }
    else
    {
        i_up = i + 1;
        i_down = i - 1;
    }
    if ( j == 0 )
    {
        j_left = length - 1;
        j_right = 1;
    }
    else if ( j == length - 1)
    {
        j_left = j - 1;
        j_right = 0;
    }
    else
    {
        j_left = j - 1;
        j_right = j + 1;
    }

    ij[0] = i_up;
    ij[1] = i_down;
    ij[2] = j_left;
    ij[3] = j_right;
}

__device__
void accept_reject( float a, float q, float r, float m, float *x1_grid,
                    float *r1_grid, int index_global )
{
    float x1 = x1_grid[ index_global ];

    x1 = a * fmod( x1, q ) - ( r * x1 ) / q;

    if ( x1 < 0 )
    {
        x1 += m;
    }

    float r1 = x1 / m;

    x1_grid[ index_global ] = x1;
    r1_grid[ index_global ] = r1;
}

__global__
void GPUKernel_update_grid( int *d_grid, int length, float J, float beta, float a,
                                float q, float r, float m, float *d_x1_grid,
                                float *d_r1_grid, float *d_y_grid, int parity )
{
    // Compute the global location of the active thread.
    int x_global = blockIdx.x * blockDim.x + threadIdx.x;
    int y_global = blockIdx.y * blockDim.y + threadIdx.y;
    int index_global = ( length * x_global ) + y_global;

    int x_local = threadIdx.x;
    int y_local = threadIdx.y;
    int index_local = ( blockDim.y * x_local ) + y_local;

    // Declare shared data.
    extern __shared__ int shared[];

    // Populate the shared data array.
    (shared)[ index_local ] = (d_grid)[ index_global ];
    // shared[ ( blockDim.y * threadIdx.x ) + threadIdx.y ] =
    //     (d_grid)[ ( length * x_global ) + y_global ];

    // Wait for all threads to finish.
    __syncthreads();

    int x_up_local, x_down_local, x_up_global, x_down_global;
    int y_left_local, y_right_local, y_left_global, y_right_global;

    int up_index_spin, down_index_spin, left_index_spin, right_index_spin;

    // float energy_old, energy_new, y, x1, r1;
    float energy_old, energy_new, y, r1;
    bool change = false;

    // If the thread is within the bounds of the grid ...
    if ( ( x_global < length ) && ( y_global < length )
        && ( ( ( x_global + y_global ) % 2 == parity ) ) )
    {
        // Compute local neighboring indices.

        // If the thread is in the first row of threads in the local grid ...
        if ( threadIdx.x == 0 )
        {
            // There is no local memory going upwards.
            x_up_local = NULL;

            // If the upward neighbor is out of bounds of the global grid ...
            if ( ( x_global - 1 ) < 0 )
            {
                // Wrap around to the last row in the global grid.
                x_up_global = length - 1;
            }
            // Otherwise ...
            else
            {
                // Access the memory in the global grid one row above.
                x_up_global = x_global - 1;
            }

            // The downward neighbor is always at row 1 in the local grid.
            x_down_local = 1;
            // The downward global memory does not need to be accessed.
            x_down_global = NULL;

            up_index_spin   = (d_grid)[ length     * x_up_global  + y_global ];
            down_index_spin = (shared)[ blockDim.y * x_down_local + y_local  ];

        }
        // Else if the thread is in the last row of threads in the local grid ...
        else if ( threadIdx.x == ( blockDim.x - 1 ) )
        {
            // The upwards neighbor is in the row above in the local grid.
            x_up_local = x_local - 1;
            // The global memory does not need to be accessed.
            x_up_global = NULL;

            // There is no local memory going downwards.
            x_down_local = NULL;

            // If the downwards neighbor is out of bounds of the global grid ...
            if ( ( x_global + 1 ) > ( length - 1 ) )
            {
                // Wrap around to the first row of the global grid.
                x_down_global = 0;
            }
            // Otherwise ...
            else
            {
                // Access the memory in the global grid one row below.
                x_down_global = x_global + 1;
            }

            up_index_spin   = (shared)[ blockDim.y * x_up_local    + y_local  ];
            down_index_spin = (d_grid)[ length     * x_down_global + y_global ];

        }
        // Else if the thread is neither in the first row nor the last row of the
        //  local grid.
        else
        {
            // The upwards neighbor is in the row above in the local grid.
            x_up_local = x_local - 1;
            // The global memory does not need to be accessed.
            x_up_global = NULL;

            // The downwards neighbor is in the row below in the local grid.
            x_down_local = x_local + 1;
            // The global memory does not need to be accessed.
            x_down_global = NULL;

            up_index_spin   = (shared)[ blockDim.y * x_up_local   + y_local ];
            down_index_spin = (shared)[ blockDim.y * x_down_local + y_local ];
        }

        // If the thread is in the first column of threads in the local grid ...
        if ( threadIdx.y == 0 )
        {
            // There is no local memory going leftwards.
            y_left_local = NULL;

            // If the leftwards neighbor is out of bounds of the global grid ...
            if ( ( y_global - 1 ) < 0 )
            {
                // Wrap around to the last column in the global grid.
                y_left_global = length - 1;
            }
            // Otherwise ...
            else
            {
                // Access the memory in the global grid one column to the left.
                y_left_global = y_global - 1;
            }

            // The rightwards neighbor is always at column 1 in the local grid.
            y_right_local = 1;
            // The global memory does not need to be accessed.
            y_right_global = NULL;

            left_index_spin  = (d_grid)[ length     * x_global + y_left_global ];
            right_index_spin = (shared)[ blockDim.y * x_local  + y_right_local ];
        }
        // Else if the thread is in the last column of threads in the local grid ...
        else if ( threadIdx.y == ( blockDim.y - 1 ) )
        {
            // The leftwards neighbor is in the column to the left in the local grid.
            y_left_local = y_local - 1;
            // The global memory does not need to be accessed.
            y_left_global = NULL;

            // There is no local memory going rightwards.
            y_right_local = NULL;

            // If the rightwards neighbor is out of bounds of the global grid ...
            if ( ( y_global + 1 ) > ( length - 1 ) )
            {
                // Wrap around to the first column of the global grid.
                y_right_global = 0;
            }
            // Otherwise ...
            else
            {
                // Access the memory in the global grid one column to the right.
                y_right_global = y_global + 1;
            }

            left_index_spin  = (shared)[ blockDim.y * x_local  + y_left_local   ];
            right_index_spin = (d_grid)[ length     * x_global + y_right_global ];

        }
        // Else if the thread is neither in the first column nor the last column
        //  of the local grid.
        else
        {
            // The leftwards neighbor is in the column to the left in the local grid.
            y_left_local = y_local - 1;
            // The global memory does not need to be accessed.
            y_left_global = NULL;

            // The rightwards neighbor is in the column to the right in the local grid.
            y_right_local = y_local + 1;
            // The global memory does not need to be accessed.
            y_right_global = NULL;

            left_index_spin  = (shared)[ blockDim.y * x_local + y_left_local  ];
            right_index_spin = (shared)[ blockDim.y * x_local + y_right_local ];
        }

        // printf("up_spin: %d, down_spin: %d, left_spin: %d, right_spin: %d\n",
        //         up_index_spin, down_index_spin, left_index_spin, right_index_spin );

        energy_old = -J * (shared)[ index_local ] * ( up_index_spin + down_index_spin
            + left_index_spin + right_index_spin );

        energy_new = - energy_old;

        if ( energy_new <= energy_old )
        {
            change = true;
        }
        else
        {
            // printf( "energy_old: %f, energy_new: %f\n", energy_old, energy_new );

            y = exp( -beta * ( energy_new - energy_old ) );

            d_y_grid[ index_global ] = y;

            accept_reject( a, q, r, m, d_x1_grid, d_r1_grid, index_global );

            r1 = d_r1_grid[ index_global ];

            // printf( "r1: %f, y: %f\n", r1, y );

            if ( r1 <= y )
            {
                change = true;
            }
            else
            {
                change = false;
            }
        }

        if ( change == true )
        {
            (d_grid)[ index_global ] = -(d_grid)[ index_global ];
        }
    }

}

// void update_grid( int *grid, int length, float J, float beta, float a, float q,
//                         float r, float m, float *x1_grid, float *r1_grid )
// {
//     int i, j, i_up, i_down, j_left, j_right;
//     int index, up_index, down_index, left_index, right_index;
//     float energy_old, energy_new, y, r1;
//     bool change;
//
//     for ( i = 0; i < length; i++ )
//     {
//         for ( j = 0; j < length; j++ )
//         {
//             index = ( length * i ) + j;
//
//             if ( i == 0 )
//             {
//                 i_up = 1;
//                 i_down = length - 1;
//             }
//             else if ( i == length - 1)
//             {
//                 i_up = 0;
//                 i_down = i - 1;
//             }
//             else
//             {
//                 i_up = i + 1;
//                 i_down = i - 1;
//             }
//             if ( j == 0 )
//             {
//                 j_left = length - 1;
//                 j_right = 1;
//             }
//             else if ( j == length - 1)
//             {
//                 j_left = j - 1;
//                 j_right = 0;
//             }
//             else
//             {
//                 j_left = j - 1;
//                 j_right = j + 1;
//             }
//
//             up_index    = ( length * i_up )   + j;
//             down_index  = ( length * i_down ) + j;
//             left_index  = ( length * i )      + j_left;
//             right_index = ( length * i )      + j_right;
//
//             energy_old = -J * (grid)[ index ] * ( (grid)[ up_index ] + (grid)[ down_index ]
//                 + (grid)[ left_index ] + (grid)[ right_index ] );
//
//             energy_new = - energy_old;
//
//             if ( energy_new <= energy_old )
//             {
//                 change = true;
//             }
//             else
//             {
//                 y = exp( -beta * ( energy_new - energy_old ) );
//                 accept_reject( y, a, q, r, m, x1_grid, r1_grid, index );
//
//                 r1 = r1_grid[ index ];
//
//                 if ( r1 <= y )
//                 {
//                     change = true;
//                 }
//                 else
//                 {
//                     change = false;
//                 }
//             }
//
//             if ( change == true )
//             {
//                 (grid)[ index ] = -(grid)[ index ];
//             }
//         }
//     }
// }



int main( int argc, char *argv[] )
{
    if ( argc != 6 )
    {
        // Print out the necessary command line imputs.
        printf( "Arguments for execution: %s <filename> <length> <J> <beta> <trajecs> <blocksize>\n", argv[0] );
    }

    int length = std::stoi( argv[1] );
    int size = length * length;

    float J = std::stof( argv[2] );
    float beta = std::stof( argv[3] );
    int trajecs = std::stoi( argv[4] );

    int blockwidth = std::stoi( argv[5] );
    // The number of thread blocks in the x, y, and z directions.
    dim3 dimGrid( length/blockwidth, length/blockwidth, 1 );
    // The number of threads in the x, y, and z directions of a thread block.
    dim3 dimBlock( blockwidth, blockwidth, 1 );

    float a = pow( 7., 5 );
    float m = pow( 2., 31 ) - 1.;

    float q = m / a;
    float r = fmod( m, a );

    int *h_grid;
    h_grid = (int *)malloc( sizeof(int) * size );
    int *d_grid;
    hipMalloc( (void **)&d_grid, sizeof(int) * size );

    float *h_x1_grid;
    h_x1_grid = (float *)malloc( sizeof(float) * size );
    float *d_x1_grid;
    hipMalloc( (void **)&d_x1_grid, sizeof(float) * size );

    float *h_r1_grid;
    h_r1_grid = (float *)malloc( sizeof(float) * size );
    float *d_r1_grid;
    hipMalloc( (void **)&d_r1_grid, sizeof(float) * size );

    // float *h_y_grid;
    // h_y_grid = (float *)malloc( sizeof(float) * size );
    float *d_y_grid;
    hipMalloc( (void **)&d_y_grid, sizeof(float) * size );

    initialize_x1_grid( a, q, r, m, h_x1_grid, length );
    initialize_grid( h_grid, length );
    // print_grid( h_grid, length, 0 );

    hipMemcpy( d_grid, h_grid, sizeof(int) * size, hipMemcpyHostToDevice );
    hipMemcpy( d_x1_grid, h_x1_grid, sizeof(float) * size, hipMemcpyHostToDevice );
    hipMemcpy( d_r1_grid, h_r1_grid, sizeof(float) * size, hipMemcpyHostToDevice );

    // print_other_grid( h_x1_grid, length, 0, "x1" );

    // float *d2h_x1_grid;
    // d2h_x1_grid = (float *)malloc( sizeof(float) * size );

    // cudaMemcpy( d2h_x1_grid, d_x1_grid, sizeof(int) * size, cudaMemcpyDeviceToHost );

    // print_other_grid( d2h_x1_grid, length, 1, "x1");

    struct timeval comp_start, comp_end;
    gettimeofday( &comp_start, NULL );

    for ( int t = 1; t < trajecs; t++ )
    {
        GPUKernel_update_grid<<< dimGrid, dimBlock, sizeof(int) * blockwidth * blockwidth >>>
            ( d_grid, length, J, beta, a, q, r, m, d_x1_grid, d_r1_grid, d_y_grid, 0 );
        // cudaDeviceSynchronize();

        // cudaMemcpy( h_grid, d_grid, sizeof(int) * size, cudaMemcpyDeviceToHost );
        // print_grid( h_grid, length, t*2 );

        GPUKernel_update_grid<<< dimGrid, dimBlock, sizeof(int) * blockwidth * blockwidth >>>
            ( d_grid, length, J, beta, a, q, r, m, d_x1_grid, d_r1_grid, d_y_grid, 1 );
        // cudaDeviceSynchronize();

        // cudaMemcpy( h_grid, d_grid, sizeof(int) * size, cudaMemcpyDeviceToHost );
        // print_grid( h_grid, length, t*2 + 1 );
        // print_grid( h_grid, length, t );

        // cudaMemcpy( d2h_x1_grid, d_x1_grid, sizeof(int) * size, cudaMemcpyDeviceToHost );
        // print_other_grid( d2h_x1_grid, length, t, "x1" );
        //
        // cudaMemcpy( h_r1_grid, d_r1_grid, sizeof(int) * size, cudaMemcpyDeviceToHost );
        // print_other_grid( h_r1_grid, length, t, "r1" );
        //
        // cudaMemcpy( h_y_grid, d_y_grid, sizeof(int) * size, cudaMemcpyDeviceToHost );
        // print_other_grid( h_y_grid, length, t, "y" );
    }

    hipDeviceSynchronize();


    // cudaMemcpy( h_grid, d_grid, sizeof(int) * size, cudaMemcpyDeviceToHost );
    //
    // print_grid( h_grid, length, 999999 );

    gettimeofday( &comp_end, NULL );

    float comp_time_microseconds = ( comp_end.tv_sec * 1000000 + comp_end.tv_usec )
        - ( comp_start.tv_sec * 1000000 + comp_start.tv_usec );
    float comp_time_milliseconds = comp_time_microseconds / 1000.;

    printf( "%d, %f, %f, %d, %d, ", length, J, beta, trajecs, blockwidth );
    printf( "%lf\n", comp_time_milliseconds );



}
