// ################################################################################################## //
//                                     ising_model.cu                                                 //
// ################################################################################################## //
/*
////////////////////////////////////////////////////////////////////////////////////////////////////////
// Johnson Liu ( j_liu21@u.pacific.edu ) 20210422                                                     //
// University of the Pacific                                                                          //
// CA USA                                                                                             //
////////////////////////////////////////////////////////////////////////////////////////////////////////
########################################################################################################
--------------------------------------------------------------------------------------------------------
Functions of the code: ---------------------------------------------------------------------------------
    1)

--------------------------------------------------------------------------------------------------------
Notes: -------------------------------------------------------------------------------------------------
    1)

--------------------------------------------------------------------------------------------------------
Input: -------------------------------------------------------------------------------------------------
    1)

--------------------------------------------------------------------------------------------------------
Output: ------------------------------------------------------------------------------------------------
    A) To screen:
        1)

    B) Printed to ... :
        1)
--------------------------------------------------------------------------------------------------------
########################################################################################################
--------------------------------------------------------------------------------------------------------
1st Update

--------------------------------------------------------------------------------------------------------

########################################################################################################
*/
// =====================================================================================================
// Import libraries to use various functions.

#include <hip/hip_runtime.h>
#include <time.h>                   // For measuring runtimes.
#include <sys/time.h>               // For measuring runtimes.
#include <stdlib.h>
#include <stdio.h>
#include <iostream>                 // For printing to screen.
#include <math.h>                   // For sqrt and atan2.
#include <unistd.h>
#include <fstream>
// #include <cuda.h>                   // For CUDA parallelization on GPU.
// =====================================================================================================


void initialize_lattice( int *grid, int length )
{
    int i, j, index;
    float r;
    for ( i = 0; i < length; i++ )
    {
        for ( j = 0; j < length; j++ )
        {
            r = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
            index = length * i + j;

            if ( r <= 0.5 )
            {
                (grid)[ index ] = -1;
            }
            else
            {
                (grid)[ index ] = 1;
            }
        }
    }
}

void print_lattice( int *grid, int length, int t )
{
    int i, j, index, spin;

    // std::string new_string = std::string(n_zero - old_string.length(), '0') + old_string;

    // std::string filename = "data/20210423_001_grid_t_" + std::to_string( t ) + ".out";

    // std::string filename = "grid_t_" + std::to_string( t ) + ".out";

    int n_zero = 5;
    std::string num_string = std::to_string( t );
    // std::string padded_num_string = std::string( n_zero - num_string.length(), '0' ) + num_string;
    // std::string filename = "grid_t_" + padded_num_string;
    // std::ofstream outfile ( num_string );

    int string_length = num_string.length();
    for ( int k = 0; k < n_zero - string_length; k++ )
    {
        num_string = "0" + num_string;
    }

    std::string filename = "grid_t_" + num_string + ".out";
    std::ofstream outfile ( filename );

    for ( i = 0; i < length; i++)
    {
        for ( j = 0; j < length; j++ )
        {
            index = length * i + j;
            spin = (grid)[ index ];

            if ( spin == 1)
            {
                // std::cout << " " << 1 <<  ", ";
                outfile << " " << 1 <<  ", ";
            }
            else
            {
                // std::cout << -1 <<  ", ";
                outfile << -1 <<  ", ";
            }

        }
        // std::cout << std::endl;
        outfile << std::endl;
    }

    outfile.close();
}

void determine_ij( int i, int j, int length, int *ij )
{
    int i_up, i_down, j_left, j_right;

    if ( i == 0 )
    {
        i_up = 1;
        i_down = length - 1;
    }
    else if ( i == length - 1)
    {
        i_up = 0;
        i_down = i - 1;
    }
    else
    {
        i_up = i + 1;
        i_down = i - 1;
    }
    if ( j == 0 )
    {
        j_left = length - 1;
        j_right = 1;
    }
    else if ( j == length - 1)
    {
        j_left = j - 1;
        j_right = 0;
    }
    else
    {
        j_left = j - 1;
        j_right = j + 1;
    }

    // int *ij;
    // ij = (int *)malloc( sizeof(int) * 4 );
    ij[0] = i_up;
    ij[1] = i_down;
    ij[2] = j_left;
    ij[3] = j_right;
}

void accept_reject( float y, float a, float q, float r, float m, float *x1r1 )
{

    float x1 = x1r1[0];
    x1 = a * fmod( x1, q ) - ( r * x1 ) / q;

    if ( x1 < 0 )
    {
        x1 += m;
    }

    float r1 = x1 / m;

    x1r1[0] = x1;
    x1r1[1] = r1;

    // std::cout << "x1: " << x1 << ", r1: " << r1 << std::endl;

}

void update_lattice( int *grid, int length, float J, float beta,
                        float a, float q, float r, float m, int *ij, float *x1r1 )
// void update_lattice( int *grid, int length, float J, float beta, int *ij )
{
    int i, j, index, up_index, down_index, left_index, right_index;
    // float n;
    float energy_old, energy_new, y, r1;
    // float energy_old, energy_new, y;
    bool change;

    for ( i = 0; i < length; i++ )
    {
        for ( j = 0; j < length; j++ )
        {
            index = length * i + j;
            determine_ij( i, j, length, ij );

            up_index    = length * ij[0] + j;
            down_index  = length * ij[1] + j;
            left_index  = length * i + ij[2];
            right_index = length * i + ij[3];

            energy_old = -J * (grid)[ index ] * ( (grid)[ up_index ] + (grid)[ down_index ]
                + (grid)[ left_index ] + (grid)[ right_index ] );

            energy_new = - energy_old;

            if ( energy_new <= energy_old )
            {
                change = true;
            }
            else
            {
                y = exp( -beta * ( energy_new - energy_old ) );
                accept_reject( y, a, q, r, m, x1r1 );
                // n = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);

                // x1 = x1r1[0];
                r1 = x1r1[1];

                // std::cout << "y: " << y << ", r1: " << r1 << std::endl;

                if ( r1 <= y )
                // if ( n <= y )
                {
                    change = true;
                }
                else
                {
                    change = false;
                }
            }

            if ( change == true )
            {
                (grid)[ index ] = -(grid)[ index ];
            }

            // std::cout << "up: " << ij[0] << ", down: " << ij[1] << ", left: " << ij[2] << ", right: " << ij[3] << std::endl;
        }
    }
}



int main( int argc, char *argv[] )
{
    if ( argc != 5 )
    {
        // Print out the necessary command line imputs.
        printf( "Arguments for execution: %s <filename> <length> <J> <beta> <trajecs>\n", argv[0] );
    }

    int length = std::stoi( argv[1] );
    int size = length * length;

    float J = std::stof( argv[2] );
    float beta = std::stof( argv[3] );
    int trajecs = std::stoi( argv[4] );

    int *grid;
    grid = (int *)malloc( sizeof(int) * size );

    int *ij;
    ij = (int *)malloc( sizeof(int) * 4 );

    float *x1r1;
    x1r1 = (float *)malloc( sizeof(float) * 2 );

    float a = pow( 7., 5 );
    float m = pow( 2., 31 ) - 1.;

    float q = m / a;
    float r = fmod( m, a );

    float x1 = pow( 5.5, 13 );
    x1r1[0] = x1;

    // std::cout << "a: " << a << ", m: " << m << ", q: " << q << ", r: " << r << ", x1: " << x1 << std::endl;

    // std::cout << "fmod(3.5, 2): " << fmod(3.5, 2) << std::endl;

    initialize_lattice( grid, length );
    // print_lattice( grid, length, 0 );

    /*
    struct timeval comp_start, comp_end;
    gettimeofday( &comp_start, NULL );
    */

    for ( int t = 1; t < trajecs; t++ )
    {
        update_lattice( grid, length, J, beta, a, q, r, m, ij, x1r1 );
        print_lattice( grid, length, t );
    }

    /*
    gettimeofday( &comp_end, NULL );

    float comp_time_microseconds = ( comp_end.tv_sec * 1000000 + comp_end.tv_usec )
        - ( comp_start.tv_sec * 1000000 + comp_start.tv_usec );
    float comp_time_milliseconds = comp_time_microseconds / 1000.;

    printf( "%d, %f, %f, %d, ", length, J, beta, trajecs );
    printf( "%lf\n", comp_time_milliseconds );
    */







}
